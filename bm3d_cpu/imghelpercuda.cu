#include "imghelpercuda.h"
#include <vector>
#include <stdio.h>

void ImgHelperCuda::transform2D(hipfftReal* image, int x, int y)
{
    hipfftReal *in_h, *in_d;
    hipfftComplex *mid_d, *mid_h;
    hipfftHandle pF;

    //in_h = (hipfftReal*) malloc(x* y * sizeof(hipfftReal));
    //memcpy(in_h, image, x * y * sizeof(hipfftReal));
    in_h = image;


    //out_h= (hipfftReal*) malloc(x * y * sizeof(hipfftReal));
    mid_h= (hipfftComplex*)malloc(x * y *sizeof(hipfftComplex));

    hipMalloc((void**) &in_d, x * y * sizeof(hipfftReal));
    hipMalloc((void**)&mid_d, x * y * sizeof(hipfftComplex));

    hipfftPlan2d(&pF, x , y, HIPFFT_R2C);

    hipMemcpy((hipfftReal*)in_d, (hipfftReal*)in_h, x * y * sizeof(hipfftReal),hipMemcpyHostToDevice);

    hipfftExecR2C(pF, (hipfftReal*)in_d, (hipfftComplex*)mid_d);

    hipMemcpy((hipfftComplex*)mid_h, (hipfftComplex*)mid_d, y * x * sizeof(hipfftComplex), hipMemcpyDeviceToHost);

    for(int i=0; i< x * y; ++i)
    {
        image[i] = mid_h[i].x;
        //printf("%f,", image[i]);
    }
}
