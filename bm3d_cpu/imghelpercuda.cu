#include "hip/hip_runtime.h"
#include "imghelpercuda.h"
#include <vector>
#include <stdio.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hipfft/hipfft.h>

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void ImgHelperCuda::gpuAssert(hipError_t code, const char *file, int line, bool abort)
{
   if (code != hipSuccess)
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

void ImgHelperCuda::CheckCufftError(hipfftResult result, const char* method)
{
    switch(result)
    {
        case HIPFFT_SUCCESS:
            //printf("\n CUFFT (%s): SUCCESS", method);
            break;
        case HIPFFT_ALLOC_FAILED:
            printf("\n CUFFT (%s): Allocation failed", method);
            break;
        case HIPFFT_INVALID_VALUE:
            printf("\n CUFFT (%s): Invalid value", method);
            break;
        case HIPFFT_INTERNAL_ERROR:
            printf("\n CUFFT (%s): Internal error", method);
            break;
        case HIPFFT_SETUP_FAILED:
            printf("\n CUFFT (%s): Setup failed", method);
            break;
        case HIPFFT_INVALID_SIZE:
            printf("\n CUFFT (%s): Invalid size", method);
            break;
        default:
            printf("\n CUFFT (%s): unkown error", method);
    };
}

void ImgHelperCuda::fft(float* src, hipfftComplex* dst, int width, int height)
{
    float* plainSrc;
    hipfftComplex* plainDst;

    gpuErrchk(hipMalloc(&plainSrc, width * height * sizeof(float)));
    gpuErrchk(hipMalloc(&plainDst,  width * ((height/2) + 1) * sizeof(hipfftComplex)));
    gpuErrchk(hipMemcpy(plainSrc,src,width * height * sizeof(float),hipMemcpyHostToDevice));

    hipfftHandle handle;
    hipfftResult r = hipfftPlan2d(&handle,width,height,HIPFFT_R2C);
    CheckCufftError(r, "hipfftPlan2d");

    r = cufftSetCompatibilityMode(handle,CUFFT_COMPATIBILITY_NATIVE);
    CheckCufftError(r, "cufftSetCompatibilityMode");

    r = hipfftExecR2C(handle,plainSrc,plainDst);
    CheckCufftError(r, "hipfftExecR2C");
    hipDeviceSynchronize ();

    gpuErrchk(hipMemcpy(dst, plainDst, width * ((height/2) + 1) * sizeof(hipfftComplex),hipMemcpyDeviceToHost));

    r = hipfftDestroy(handle);
    CheckCufftError(r, "hipfftDestroy");

    hipFree(plainSrc);
    hipFree(plainDst);
}

hipfftComplex* ImgHelperCuda::fft2(float* src, int width, int height)
{
    float* plainSrc;
    hipfftComplex* plainDst;

    gpuErrchk(hipMalloc(&plainSrc, width * height * sizeof(float)));
    gpuErrchk(hipMalloc(&plainDst,  width * ((height/2) + 1) * sizeof(hipfftComplex)));
    gpuErrchk(hipMemcpy(plainSrc,src,width * height * sizeof(float),hipMemcpyHostToDevice));

    hipfftHandle handle;
    hipfftResult r = hipfftPlan2d(&handle,width,height,HIPFFT_R2C);
    CheckCufftError(r, "hipfftPlan2d");

    r = cufftSetCompatibilityMode(handle,CUFFT_COMPATIBILITY_NATIVE);
    CheckCufftError(r, "cufftSetCompatibilityMode");

    r = hipfftExecR2C(handle,plainSrc,plainDst);
    CheckCufftError(r, "hipfftExecR2C");
    hipDeviceSynchronize ();

    //gpuErrchk(hipMemcpy(dst, plainDst, width * ((height/2) + 1) * sizeof(hipfftComplex),hipMemcpyDeviceToHost));

    r = hipfftDestroy(handle);
    CheckCufftError(r, "hipfftDestroy");

    //hipFree(plainSrc);
    //hipFree(plainDst);
    return plainDst;
}

void ImgHelperCuda::ifft(hipfftComplex* src, float* dst, int width, int height)
{
    hipfftComplex* plainSrc;
    float* plainDst;

    gpuErrchk(hipMalloc(&plainSrc, width * ((height/2) + 1) * sizeof(hipfftComplex)));
    gpuErrchk(hipMalloc(&plainDst, width * height * sizeof(float)));
    gpuErrchk(hipMemcpy(plainSrc,src,width * ((height/2) + 1) * sizeof(hipfftComplex),hipMemcpyHostToDevice));

    hipfftHandle handle;
    hipfftResult r = hipfftPlan2d(&handle,width,height,HIPFFT_C2R);
    CheckCufftError(r, "hipfftPlan2d");

    r = cufftSetCompatibilityMode(handle,CUFFT_COMPATIBILITY_NATIVE);
    CheckCufftError(r, "cufftSetCompatibilityMode");

    r = hipfftExecC2R(handle,plainSrc,plainDst);
    CheckCufftError(r, "hipfftExecZ2D");
    hipDeviceSynchronize ();

    gpuErrchk(hipMemcpy(dst,plainDst,width * height * sizeof(float),hipMemcpyDeviceToHost));

    r = hipfftDestroy(handle);
    CheckCufftError(r, "hipfftDestroy");

    hipFree(plainSrc);
    hipFree(plainDst);
}

float* ImgHelperCuda::ifft2(hipfftComplex* src, int width, int height)
{
    hipfftComplex* plainSrc;
    float* plainDst;

    //gpuErrchk(hipMalloc(&plainSrc, width * ((height/2) + 1) * sizeof(hipfftComplex)));
    gpuErrchk(hipMalloc(&plainDst, width * height * sizeof(float)));
    //gpuErrchk(hipMemcpy(plainSrc,src,width * ((height/2) + 1) * sizeof(hipfftComplex),hipMemcpyHostToDevice));

    hipfftHandle handle;
    hipfftResult r = hipfftPlan2d(&handle,width,height,HIPFFT_C2R);
    CheckCufftError(r, "hipfftPlan2d");

    r = cufftSetCompatibilityMode(handle,CUFFT_COMPATIBILITY_NATIVE);
    CheckCufftError(r, "cufftSetCompatibilityMode");

    r = hipfftExecC2R(handle,src,plainDst);
    CheckCufftError(r, "hipfftExecZ2D");
    hipDeviceSynchronize ();

    float* dst = (float*)malloc(width * height * sizeof(float));
    gpuErrchk(hipMemcpy(dst,plainDst,width * height * sizeof(float),hipMemcpyDeviceToHost));

    r = hipfftDestroy(handle);
    CheckCufftError(r, "hipfftDestroy");

    //hipFree(plainSrc);
    //hipFree(plainDst);

    return dst;
}

__global__
void Process2DHT_intern(hipfftComplex* src, int gamma)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x + blockIdx.y * blockDim.y + threadIdx.y;
    if(src[i].x <= gamma) src[i].x = 0;
}


void ImgHelperCuda::Process2DHT(hipfftComplex* src, int gamma)
{
    dim3 blocks(5,5);
    dim3 threads(8,8);
    Process2DHT_intern<<<blocks,threads>>>(src, gamma);
    hipDeviceSynchronize ();
}




