#include "imghelpercuda.h"
#include <vector>
#include <stdio.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hipfft/hipfft.h>

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void ImgHelperCuda::gpuAssert(hipError_t code, const char *file, int line, bool abort)
{
   if (code != hipSuccess)
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

void ImgHelperCuda::CheckCufftError(hipfftResult result, const char* method)
{
    switch(result)
    {
        case HIPFFT_SUCCESS:
            //printf("\n CUFFT (%s): SUCCESS", method);
            break;
        case HIPFFT_ALLOC_FAILED:
            printf("\n CUFFT (%s): Allocation failed", method);
            break;
        case HIPFFT_INVALID_VALUE:
            printf("\n CUFFT (%s): Invalid value", method);
            break;
        case HIPFFT_INTERNAL_ERROR:
            printf("\n CUFFT (%s): Internal error", method);
            break;
        case HIPFFT_SETUP_FAILED:
            printf("\n CUFFT (%s): Setup failed", method);
            break;
        case HIPFFT_INVALID_SIZE:
            printf("\n CUFFT (%s): Invalid size", method);
            break;
        default:
            printf("\n CUFFT (%s): unkown error", method);
    };
}

void ImgHelperCuda::fft_device_double(double* src, hipfftDoubleComplex* dst, int width, int height)
{
    double* plainSrc;
    hipfftDoubleComplex* plainDst;

    gpuErrchk(hipMalloc(&plainSrc, width * height * sizeof(double)));
    gpuErrchk(hipMalloc(&plainDst,  width * ((height/2) + 1) * sizeof(hipfftDoubleComplex)));
    gpuErrchk(hipMemcpy(plainSrc,src,width * height * sizeof(double),hipMemcpyHostToDevice));

    hipfftHandle handle;
    hipfftResult r = hipfftPlan2d(&handle,width,height,HIPFFT_D2Z);
    CheckCufftError(r, "hipfftPlan2d");

    r = cufftSetCompatibilityMode(handle,CUFFT_COMPATIBILITY_NATIVE);
    CheckCufftError(r, "cufftSetCompatibilityMode");

    r = hipfftExecD2Z(handle,plainSrc,plainDst);
    CheckCufftError(r, "hipfftExecD2Z");
    hipDeviceSynchronize ();

    gpuErrchk(hipMemcpy(dst, plainDst, width * ((height/2) + 1) * sizeof(hipfftDoubleComplex),hipMemcpyDeviceToHost));

    r = hipfftDestroy(handle);
    CheckCufftError(r, "hipfftDestroy");

    hipFree(plainSrc);
    hipFree(plainDst);
}

void ImgHelperCuda::fft_inverse_device_double(hipfftDoubleComplex* src, double* dst, int width, int height)
{
    hipfftDoubleComplex* plainSrc;
    double* plainDst;

    gpuErrchk(hipMalloc(&plainSrc, width * ((height/2) + 1) * sizeof(hipfftDoubleComplex)));
    gpuErrchk(hipMalloc(&plainDst, width * height * sizeof(double)));
    gpuErrchk(hipMemcpy(plainSrc,src,width * ((height/2) + 1) * sizeof(hipfftDoubleComplex),hipMemcpyHostToDevice));

    hipfftHandle handle;
    hipfftResult r = hipfftPlan2d(&handle,width,height,HIPFFT_Z2D);
    CheckCufftError(r, "hipfftPlan2d");

    r = cufftSetCompatibilityMode(handle,CUFFT_COMPATIBILITY_NATIVE);
    CheckCufftError(r, "cufftSetCompatibilityMode");

    r = hipfftExecZ2D(handle,plainSrc,plainDst);
    CheckCufftError(r, "hipfftExecZ2D");
    hipDeviceSynchronize ();

    gpuErrchk(hipMemcpy(dst,plainDst,width * height * sizeof(double),hipMemcpyDeviceToHost));

    r = hipfftDestroy(handle);
    CheckCufftError(r, "hipfftDestroy");

    hipFree(plainSrc);
    hipFree(plainDst);
}

void ImgHelperCuda::fft_device(float* src, hipfftComplex* dst, int width, int height)
{
    //src and dst are device pointers allocated with hipMallocPitch

    //Convert them to plain pointers. No padding of rows.
    for(int i = 0; i < 10; ++i)
    {
        printf("%f,", src[i]);
    }
    printf("\n");

    float* plainSrc;
    hipfftComplex* plainDst;

    gpuErrchk(hipMalloc<float>(&plainSrc,width * height * sizeof(float)));
    gpuErrchk(hipMalloc<hipfftComplex>(&plainDst, width * height * sizeof(hipfftComplex)));
    gpuErrchk(hipMemcpy(plainSrc,src,width * height * sizeof(float),hipMemcpyHostToDevice));

    hipfftHandle handle;
    hipfftResult r = hipfftPlan2d(&handle,width,height,HIPFFT_R2C);
    CheckCufftError(r, "hipfftPlan2d");

    r = cufftSetCompatibilityMode(handle,CUFFT_COMPATIBILITY_NATIVE);
    CheckCufftError(r, "cufftSetCompatibilityMode");

    r = hipfftExecR2C(handle,plainSrc,plainDst);
    CheckCufftError(r, "hipfftExecR2C");
    hipDeviceSynchronize ();

    gpuErrchk(hipMemcpy(dst,plainDst,width * height * sizeof(hipfftComplex),hipMemcpyDeviceToHost));

    r = hipfftDestroy(handle);
    CheckCufftError(r, "hipfftDestroy");

    hipFree(plainSrc);
    hipFree(plainDst);

    for(int i = 0; i < 10; ++i)
    {
        printf("%f,", dst[i].x);
    }
    printf("\n");
}

void ImgHelperCuda::fft_inverse_device(hipfftComplex* src, float* dst, int width, int height)
{
    //src and dst are device pointers allocated with hipMallocPitch

    //Convert them to plain pointers. No padding of rows.
    for(int i = 0; i < 10; ++i)
    {
        printf("%f,", src[i].x);
    }
    printf("\n");

    hipfftComplex* plainSrc;
    float* plainDst;

    gpuErrchk(hipMalloc<hipfftComplex>(&plainSrc, width * height * sizeof(hipfftComplex)));
    gpuErrchk(hipMalloc<float>(&plainDst, width * height * sizeof(float)));
    gpuErrchk(hipMemcpy(plainSrc,src,width * height * sizeof(hipfftComplex),hipMemcpyHostToDevice));

    hipfftHandle handle;
    hipfftResult r = hipfftPlan2d(&handle,width,height,HIPFFT_C2R);
    CheckCufftError(r, "hipfftPlan2d");

    r = cufftSetCompatibilityMode(handle,CUFFT_COMPATIBILITY_NATIVE);
    CheckCufftError(r, "cufftSetCompatibilityMode");

    r = hipfftExecC2R(handle,plainSrc,plainDst);
    CheckCufftError(r, "hipfftExecC2R");
    hipDeviceSynchronize ();

    gpuErrchk(hipMemcpy(dst,plainDst,width * height * sizeof(float),hipMemcpyDeviceToHost));

    r = hipfftDestroy(handle);
    CheckCufftError(r, "hipfftDestroy");

    hipFree(plainSrc);
    hipFree(plainDst);

    for(int i = 0; i < 10; ++i)
    {
        printf("%f,", dst[i]);
    }
    printf("\n");
}

hipfftReal* ImgHelperCuda::InverseTransform2DTest(hipfftComplex* data, int x, int y)
{
    for(int i = 0; i < 5; ++i)
    {
        printf("%f,", data[i].x);
    }
    printf("\n");

    hipfftReal* mid_h= (hipfftReal*)malloc( x * y *sizeof(hipfftReal));
    hipfftComplex* in_d;
    hipfftReal* mid_d;
    gpuErrchk(hipMalloc((void**) &in_d, x * y * sizeof(hipfftComplex)));
    gpuErrchk(hipMalloc((void**)&mid_d, x * y * sizeof(hipfftReal)));
    gpuErrchk(hipMemcpy((hipfftComplex*)in_d, (hipfftComplex*)data, x * y * sizeof(hipfftComplex),hipMemcpyHostToDevice));

    hipfftHandle handle;
    int rank = 2; // 2D fft
    int n[] = {x, y};    // Size of the Fourier transform
    int istride = 1, ostride = 1; // Stride lengths
    int idist = 1, odist = 1;     // Distance between batches
    int inembed[] = {x, y}; // Input size with pitch
    int onembed[] = {x, y}; // Output size with pitch
    int batch = 1;
    hipfftResult r = hipfftPlanMany(&handle, rank, n,
                  inembed, istride, idist,
                  onembed, ostride, odist, HIPFFT_C2R, batch);
    CheckCufftError(r, "hipfftPlanMany");

    r = cufftSetCompatibilityMode(handle,CUFFT_COMPATIBILITY_NATIVE);
    CheckCufftError(r, "cufftSetCompatibilityMode");

    r = hipfftExecC2R(handle,data,mid_d);
    CheckCufftError(r, "hipfftExecC2R");

    gpuErrchk(hipMemcpy((hipfftReal*)mid_h, (hipfftReal*)mid_d, x * y * sizeof(hipfftReal), hipMemcpyDeviceToHost));
    for(int i = 0; i < 5; ++i)
    {
        printf("%f,", mid_h[i]);
    }
    printf("\n");

    r = hipfftDestroy(handle);
    CheckCufftError(r, "hipfftDestroy");

    hipFree(in_d);
    hipFree(mid_d);

    return mid_h;
}

hipfftComplex* ImgHelperCuda::Transform2DTest(hipfftReal* data, int x, int y)
{
    for(int i = 0; i < 5; ++i)
    {
        printf("%f,", data[i]);
    }
    printf("\n");

    size_t hostSize = x * y * sizeof(hipfftComplex);
    hipfftComplex* mid_h= (hipfftComplex*)malloc( hostSize );
    hipfftReal* in_d;
    hipfftComplex* mid_d;
    gpuErrchk(hipMalloc((void**) &in_d, x * y * sizeof(hipfftReal)));
    gpuErrchk(hipMalloc((void**)&mid_d, x * y * sizeof(hipfftComplex)));
    gpuErrchk(hipMemcpy((hipfftReal*)in_d, (hipfftReal*)data, x * y * sizeof(hipfftReal),hipMemcpyHostToDevice));

    hipfftHandle handle;
    int rank = 2; // 2D fft
    int n[] = {x, y};    // Size of the Fourier transform
    int istride = 1, ostride = 1; // Stride lengths
    int idist = 1, odist = 1;     // Distance between batches
    int inembed[] = {0}; // Input size with pitch
    int onembed[] = {0}; // Output size with pitch
    int batch = 1;
    hipfftResult r;
    /*
    r = hipfftPlanMany(&handle, rank, n,
                  inembed, istride, idist,
                  onembed, ostride, odist, HIPFFT_R2C, batch);
    CheckCufftError(r, "hipfftPlanMany");
    */

    hipfftPlan2d(&handle, x , y, HIPFFT_R2C);

    //r = cufftSetCompatibilityMode(handle,CUFFT_COMPATIBILITY_NATIVE);
    //CheckCufftError(r, "cufftSetCompatibility");

    r = hipfftExecR2C(handle, data, mid_d);
    CheckCufftError(r, "hipfftExecR2C");

    gpuErrchk(hipMemcpy((hipfftComplex*)mid_h, (hipfftComplex*)mid_d, x * y * sizeof(hipfftComplex), hipMemcpyDeviceToHost));
    for(int i = 0; i < 5; ++i)
    {
        printf("%f,", mid_h[i].x);
    }
    printf("\n");

    r = hipfftDestroy(handle);
    CheckCufftError(r, "hipfftDestroy");

    hipFree(in_d);
    hipFree(mid_d);

    return mid_h;
}

hipfftReal* ImgHelperCuda::Inversetransform2D(hipfftComplex* data, int x, int y, int* outX, int* outY)
{
    //*outX = x;
    //*outY = y;
    *outX = x;
    *outY = (y -1) * 2;
    hipfftComplex *in_d;
    hipfftReal *mid_d, *mid_h;
    hipfftHandle plan;

    for(int i = 0; i < 5; ++i)
    {
        printf("%f,", data[i].x);
    }
    printf("\n");

    mid_h= (hipfftReal*)malloc( (*outX) * (*outY) *sizeof(hipfftReal));

    hipMalloc((void**) &in_d, x * y * sizeof(hipfftComplex));
    hipMalloc((void**)&mid_d, (*outX) * (*outY) * sizeof(hipfftReal));

    hipfftPlan2d(&plan, x , y, HIPFFT_C2R);

    hipMemcpy((hipfftComplex*)in_d, (hipfftComplex*)data, x * y * sizeof(hipfftComplex),hipMemcpyHostToDevice);

    hipfftExecC2R(plan, (hipfftComplex*)in_d, (hipfftReal*)mid_d);

    hipMemcpy((hipfftReal*)mid_h, (hipfftReal*)mid_d, (*outX) * (*outY) * sizeof(hipfftReal), hipMemcpyDeviceToHost);
    for(int i = 0; i < 5; ++i)
    {
        printf("%f,", mid_h[i]);
    }
    printf("\n");

    hipfftDestroy(plan);
    hipFree(in_d);
    hipFree(mid_d);

    return mid_h;
}

hipfftComplex* ImgHelperCuda::Transform2D(hipfftReal* data, int x, int y, int* outX, int* outY)
{
    //*outX = x;
    //*outY = y;
    *outX = x;
    *outY = y/2 + 1;
    hipfftReal* in_d;
    hipfftComplex* mid_d, *mid_h;
    hipfftHandle plan;

    for(int i = 0; i < 5; ++i)
    {
        printf("%f,", data[i]);
    }
    printf("\n");

    mid_h= (hipfftComplex*)malloc( (*outX) * (*outY) *sizeof(hipfftComplex));

    hipMalloc((void**) &in_d, x * y * sizeof(hipfftReal));
    hipMalloc((void**)&mid_d, (*outX) * (*outY) * sizeof(hipfftComplex));

    hipfftPlan2d(&plan, x , y, HIPFFT_R2C);

    hipMemcpy((hipfftReal*)in_d, (hipfftReal*)data, x * y * sizeof(hipfftReal),hipMemcpyHostToDevice);

    hipfftExecR2C(plan, (hipfftReal*)in_d, (hipfftComplex*)mid_d);

    hipMemcpy((hipfftComplex*)mid_h, (hipfftComplex*)mid_d, (*outX) * (*outY) * sizeof(hipfftComplex), hipMemcpyDeviceToHost);
    for(int i = 0; i < 5; ++i)
    {
        printf("%f,", mid_h[i].x);
    }
    printf("\n");

    hipfftDestroy(plan);
    hipFree(in_d);
    hipFree(mid_d);

    return mid_h;
}
