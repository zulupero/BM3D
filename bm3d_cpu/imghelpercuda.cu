#include "imghelpercuda.h"
#include <vector>
#include <stdio.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hipfft/hipfft.h>

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void ImgHelperCuda::gpuAssert(hipError_t code, const char *file, int line, bool abort)
{
   if (code != hipSuccess)
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

void ImgHelperCuda::CheckCufftError(hipfftResult result, const char* method)
{
    switch(result)
    {
        case HIPFFT_SUCCESS:
            //printf("\n CUFFT (%s): SUCCESS", method);
            break;
        case HIPFFT_ALLOC_FAILED:
            printf("\n CUFFT (%s): Allocation failed", method);
            break;
        case HIPFFT_INVALID_VALUE:
            printf("\n CUFFT (%s): Invalid value", method);
            break;
        case HIPFFT_INTERNAL_ERROR:
            printf("\n CUFFT (%s): Internal error", method);
            break;
        case HIPFFT_SETUP_FAILED:
            printf("\n CUFFT (%s): Setup failed", method);
            break;
        case HIPFFT_INVALID_SIZE:
            printf("\n CUFFT (%s): Invalid size", method);
            break;
        default:
            printf("\n CUFFT (%s): unkown error", method);
    };
    printf("\n");
}

void ImgHelperCuda::fft_device_double(double* src, hipfftDoubleComplex* dst, int width, int height, int srcPitch, int dstPitch)
{
    //src and dst are device pointers allocated with hipMallocPitch

    //Convert them to plain pointers. No padding of rows.
    for(int i = 0; i < 10; ++i)
    {
        printf("%f,", src[i]);
    }
    printf("\n");

    double* plainSrc;
    hipfftDoubleComplex* plainDst;

    size_t pitchSrc, pitchDst;
    gpuErrchk(hipMallocPitch(&plainSrc,&pitchSrc,width * sizeof(double), height));
    gpuErrchk(hipMallocPitch(&plainDst, &pitchDst, width * sizeof(hipfftDoubleComplex), height));
    gpuErrchk(hipMemcpy2D(plainSrc,pitchSrc,src,width * sizeof(double),width * sizeof(double),height,hipMemcpyHostToDevice));
    gpuErrchk(hipMemset2D(plainSrc, pitchSrc, 0, width * sizeof(double), height));
    gpuErrchk(hipMemset2D(plainDst, pitchDst, 0, width * sizeof(hipfftDoubleComplex), height));

    hipfftHandle handle;
    hipfftResult r = hipfftPlan2d(&handle,width,height,HIPFFT_D2Z);
    CheckCufftError(r, "hipfftPlan2d");

    r = cufftSetCompatibilityMode(handle,CUFFT_COMPATIBILITY_NATIVE);
    CheckCufftError(r, "cufftSetCompatibilityMode");

    r = hipfftExecD2Z(handle,plainSrc,plainDst);
    CheckCufftError(r, "hipfftExecD2Z");
    hipDeviceSynchronize ();

    gpuErrchk(hipMemcpy2D(dst,dstPitch,plainDst,width * sizeof(hipfftDoubleComplex),width * sizeof(hipfftDoubleComplex),height,hipMemcpyDeviceToHost));

    r = hipfftDestroy(handle);
    CheckCufftError(r, "hipfftDestroy");

    hipFree(plainSrc);
    hipFree(plainDst);

    for(int i = 0; i < 10; ++i)
    {
        printf("%f,", dst[i].x);
    }
    printf("\n");
}

void ImgHelperCuda::fft_inverse_device_double(hipfftDoubleComplex* src, double* dst, int width, int height, int srcPitch, int dstPitch)
{
    //src and dst are device pointers allocated with hipMallocPitch

    //Convert them to plain pointers. No padding of rows.
    for(int i = 0; i < 10; ++i)
    {
        printf("%f,", src[i].x);
    }
    printf("\n");

    hipfftDoubleComplex* plainSrc;
    double* plainDst;

    size_t pitchSrc, pitchDst;
    gpuErrchk(hipMallocPitch(&plainSrc,&pitchSrc,width * sizeof(hipfftDoubleComplex), height));
    gpuErrchk(hipMallocPitch(&plainDst, &pitchDst, width * sizeof(double), height));
    gpuErrchk(hipMemcpy2D(plainSrc,pitchSrc,src,width * sizeof(hipfftDoubleComplex),width * sizeof(hipfftDoubleComplex),height,hipMemcpyHostToDevice));
    gpuErrchk(hipMemset2D(plainSrc, pitchSrc, 0, width * sizeof(hipfftDoubleComplex), height));
    gpuErrchk(hipMemset2D(plainDst, pitchDst, 0, width * sizeof(double), height));

    hipfftHandle handle;
    hipfftResult r = hipfftPlan2d(&handle,width,height,HIPFFT_Z2D);
    CheckCufftError(r, "hipfftPlan2d");

    r = cufftSetCompatibilityMode(handle,CUFFT_COMPATIBILITY_NATIVE);
    CheckCufftError(r, "cufftSetCompatibilityMode");

    r = hipfftExecZ2D(handle,plainSrc,plainDst);
    CheckCufftError(r, "hipfftExecZ2D");
    hipDeviceSynchronize ();

    gpuErrchk(hipMemcpy2D(dst,dstPitch,plainDst,width * sizeof(double),width * sizeof(double),height,hipMemcpyDeviceToHost));


    r = hipfftDestroy(handle);
    CheckCufftError(r, "hipfftDestroy");

    hipFree(plainSrc);
    hipFree(plainDst);

    for(int i = 0; i < 10; ++i)
    {
        printf("%f,", dst[i]);
    }
    printf("\n");
}

void ImgHelperCuda::fft_device(float* src, hipfftComplex* dst, int width, int height, int srcPitch, int dstPitch)
{
    //src and dst are device pointers allocated with hipMallocPitch

    //Convert them to plain pointers. No padding of rows.
    for(int i = 0; i < 10; ++i)
    {
        printf("%f,", src[i]);
    }
    printf("\n");

    float* plainSrc;
    hipfftComplex* plainDst;

    size_t pitchSrc, pitchDst;
    gpuErrchk(hipMallocPitch<float>(&plainSrc,&pitchSrc,width * sizeof(float), height));
    gpuErrchk(hipMallocPitch<hipfftComplex>(&plainDst, &pitchDst, width * sizeof(hipfftComplex), height));
    gpuErrchk(hipMemcpy2D(plainSrc,pitchSrc,src,width * sizeof(float),width * sizeof(float),height,hipMemcpyHostToDevice));
    gpuErrchk(hipMemset2D(plainSrc, pitchSrc, 0, width * sizeof(float), height));
    gpuErrchk(hipMemset2D(plainDst, pitchDst, 0, width * sizeof(hipfftComplex), height));

    hipfftHandle handle;
    hipfftResult r = hipfftPlan2d(&handle,width,height,HIPFFT_R2C);
    CheckCufftError(r, "hipfftPlan2d");

    r = cufftSetCompatibilityMode(handle,CUFFT_COMPATIBILITY_NATIVE);
    CheckCufftError(r, "cufftSetCompatibilityMode");

    r = hipfftExecR2C(handle,plainSrc,plainDst);
    CheckCufftError(r, "hipfftExecR2C");
    hipDeviceSynchronize ();

    gpuErrchk(hipMemcpy2D(dst,dstPitch,plainDst,width * sizeof(hipfftComplex),width * sizeof(hipfftComplex),height,hipMemcpyDeviceToHost));

    r = hipfftDestroy(handle);
    CheckCufftError(r, "hipfftDestroy");

    hipFree(plainSrc);
    hipFree(plainDst);

    for(int i = 0; i < 10; ++i)
    {
        printf("%f,", dst[i].x);
    }
    printf("\n");
}

void ImgHelperCuda::fft_inverse_device(hipfftComplex* src, float* dst, int width, int height, int srcPitch, int dstPitch)
{
    //src and dst are device pointers allocated with hipMallocPitch

    //Convert them to plain pointers. No padding of rows.
    for(int i = 0; i < 10; ++i)
    {
        printf("%f,", src[i].x);
    }
    printf("\n");

    hipfftComplex* plainSrc;
    float* plainDst;

    size_t pitchSrc, pitchDst;
    gpuErrchk(hipMallocPitch<hipfftComplex>(&plainSrc,&pitchSrc,width * sizeof(hipfftComplex), height));
    gpuErrchk(hipMallocPitch<float>(&plainDst, &pitchDst, width * sizeof(float), height));
    gpuErrchk(hipMemcpy2D(plainSrc,pitchSrc,src,width * sizeof(hipfftComplex),width * sizeof(hipfftComplex),height,hipMemcpyHostToDevice));
    gpuErrchk(hipMemset2D(plainSrc, pitchSrc, 0, width * sizeof(hipfftComplex), height));
    gpuErrchk(hipMemset2D(plainDst, pitchDst, 0, width * sizeof(float), height));

    hipfftHandle handle;
    hipfftResult r = hipfftPlan2d(&handle,width,height,HIPFFT_C2R);
    CheckCufftError(r, "hipfftPlan2d");

    r = cufftSetCompatibilityMode(handle,CUFFT_COMPATIBILITY_NATIVE);
    CheckCufftError(r, "cufftSetCompatibilityMode");

    r = hipfftExecC2R(handle,plainSrc,plainDst);
    CheckCufftError(r, "hipfftExecC2R");
    hipDeviceSynchronize ();

    gpuErrchk(hipMemcpy2D(dst,dstPitch,plainDst,width * sizeof(float),width * sizeof(float),height,hipMemcpyDeviceToHost));


    r = hipfftDestroy(handle);
    CheckCufftError(r, "hipfftDestroy");

    hipFree(plainSrc);
    hipFree(plainDst);

    for(int i = 0; i < 10; ++i)
    {
        printf("%f,", dst[i]);
    }
    printf("\n");
}

hipfftReal* ImgHelperCuda::InverseTransform2DTest(hipfftComplex* data, int x, int y)
{
    for(int i = 0; i < 5; ++i)
    {
        printf("%f,", data[i].x);
    }
    printf("\n");

    hipfftReal* mid_h= (hipfftReal*)malloc( x * y *sizeof(hipfftReal));
    hipfftComplex* in_d;
    hipfftReal* mid_d;
    gpuErrchk(hipMalloc((void**) &in_d, x * y * sizeof(hipfftComplex)));
    gpuErrchk(hipMalloc((void**)&mid_d, x * y * sizeof(hipfftReal)));
    gpuErrchk(hipMemcpy((hipfftComplex*)in_d, (hipfftComplex*)data, x * y * sizeof(hipfftComplex),hipMemcpyHostToDevice));

    hipfftHandle handle;
    int rank = 2; // 2D fft
    int n[] = {x, y};    // Size of the Fourier transform
    int istride = 1, ostride = 1; // Stride lengths
    int idist = 1, odist = 1;     // Distance between batches
    int inembed[] = {x, y}; // Input size with pitch
    int onembed[] = {x, y}; // Output size with pitch
    int batch = 1;
    hipfftResult r = hipfftPlanMany(&handle, rank, n,
                  inembed, istride, idist,
                  onembed, ostride, odist, HIPFFT_C2R, batch);
    CheckCufftError(r, "hipfftPlanMany");

    r = cufftSetCompatibilityMode(handle,CUFFT_COMPATIBILITY_NATIVE);
    CheckCufftError(r, "cufftSetCompatibilityMode");

    r = hipfftExecC2R(handle,data,mid_d);
    CheckCufftError(r, "hipfftExecC2R");

    gpuErrchk(hipMemcpy((hipfftReal*)mid_h, (hipfftReal*)mid_d, x * y * sizeof(hipfftReal), hipMemcpyDeviceToHost));
    for(int i = 0; i < 5; ++i)
    {
        printf("%f,", mid_h[i]);
    }
    printf("\n");

    r = hipfftDestroy(handle);
    CheckCufftError(r, "hipfftDestroy");

    hipFree(in_d);
    hipFree(mid_d);

    return mid_h;
}

hipfftComplex* ImgHelperCuda::Transform2DTest(hipfftReal* data, int x, int y)
{
    for(int i = 0; i < 5; ++i)
    {
        printf("%f,", data[i]);
    }
    printf("\n");

    size_t hostSize = x * y * sizeof(hipfftComplex);
    hipfftComplex* mid_h= (hipfftComplex*)malloc( hostSize );
    hipfftReal* in_d;
    hipfftComplex* mid_d;
    gpuErrchk(hipMalloc((void**) &in_d, x * y * sizeof(hipfftReal)));
    gpuErrchk(hipMalloc((void**)&mid_d, x * y * sizeof(hipfftComplex)));
    gpuErrchk(hipMemcpy((hipfftReal*)in_d, (hipfftReal*)data, x * y * sizeof(hipfftReal),hipMemcpyHostToDevice));

    hipfftHandle handle;
    int rank = 2; // 2D fft
    int n[] = {x, y};    // Size of the Fourier transform
    int istride = 1, ostride = 1; // Stride lengths
    int idist = 1, odist = 1;     // Distance between batches
    int inembed[] = {0}; // Input size with pitch
    int onembed[] = {0}; // Output size with pitch
    int batch = 1;
    hipfftResult r;
    /*
    r = hipfftPlanMany(&handle, rank, n,
                  inembed, istride, idist,
                  onembed, ostride, odist, HIPFFT_R2C, batch);
    CheckCufftError(r, "hipfftPlanMany");
    */

    hipfftPlan2d(&handle, x , y, HIPFFT_R2C);

    //r = cufftSetCompatibilityMode(handle,CUFFT_COMPATIBILITY_NATIVE);
    //CheckCufftError(r, "cufftSetCompatibility");

    r = hipfftExecR2C(handle, data, mid_d);
    CheckCufftError(r, "hipfftExecR2C");

    gpuErrchk(hipMemcpy((hipfftComplex*)mid_h, (hipfftComplex*)mid_d, x * y * sizeof(hipfftComplex), hipMemcpyDeviceToHost));
    for(int i = 0; i < 5; ++i)
    {
        printf("%f,", mid_h[i].x);
    }
    printf("\n");

    r = hipfftDestroy(handle);
    CheckCufftError(r, "hipfftDestroy");

    hipFree(in_d);
    hipFree(mid_d);

    return mid_h;
}

hipfftReal* ImgHelperCuda::Inversetransform2D(hipfftComplex* data, int x, int y, int* outX, int* outY)
{
    //*outX = x;
    //*outY = y;
    *outX = x;
    *outY = (y -1) * 2;
    hipfftComplex *in_d;
    hipfftReal *mid_d, *mid_h;
    hipfftHandle plan;

    for(int i = 0; i < 5; ++i)
    {
        printf("%f,", data[i].x);
    }
    printf("\n");

    mid_h= (hipfftReal*)malloc( (*outX) * (*outY) *sizeof(hipfftReal));

    hipMalloc((void**) &in_d, x * y * sizeof(hipfftComplex));
    hipMalloc((void**)&mid_d, (*outX) * (*outY) * sizeof(hipfftReal));

    hipfftPlan2d(&plan, x , y, HIPFFT_C2R);

    hipMemcpy((hipfftComplex*)in_d, (hipfftComplex*)data, x * y * sizeof(hipfftComplex),hipMemcpyHostToDevice);

    hipfftExecC2R(plan, (hipfftComplex*)in_d, (hipfftReal*)mid_d);

    hipMemcpy((hipfftReal*)mid_h, (hipfftReal*)mid_d, (*outX) * (*outY) * sizeof(hipfftReal), hipMemcpyDeviceToHost);
    for(int i = 0; i < 5; ++i)
    {
        printf("%f,", mid_h[i]);
    }
    printf("\n");

    hipfftDestroy(plan);
    hipFree(in_d);
    hipFree(mid_d);

    return mid_h;
}

hipfftComplex* ImgHelperCuda::Transform2D(hipfftReal* data, int x, int y, int* outX, int* outY)
{
    //*outX = x;
    //*outY = y;
    *outX = x;
    *outY = y/2 + 1;
    hipfftReal* in_d;
    hipfftComplex* mid_d, *mid_h;
    hipfftHandle plan;

    for(int i = 0; i < 5; ++i)
    {
        printf("%f,", data[i]);
    }
    printf("\n");

    mid_h= (hipfftComplex*)malloc( (*outX) * (*outY) *sizeof(hipfftComplex));

    hipMalloc((void**) &in_d, x * y * sizeof(hipfftReal));
    hipMalloc((void**)&mid_d, (*outX) * (*outY) * sizeof(hipfftComplex));

    hipfftPlan2d(&plan, x , y, HIPFFT_R2C);

    hipMemcpy((hipfftReal*)in_d, (hipfftReal*)data, x * y * sizeof(hipfftReal),hipMemcpyHostToDevice);

    hipfftExecR2C(plan, (hipfftReal*)in_d, (hipfftComplex*)mid_d);

    hipMemcpy((hipfftComplex*)mid_h, (hipfftComplex*)mid_d, (*outX) * (*outY) * sizeof(hipfftComplex), hipMemcpyDeviceToHost);
    for(int i = 0; i < 5; ++i)
    {
        printf("%f,", mid_h[i].x);
    }
    printf("\n");

    hipfftDestroy(plan);
    hipFree(in_d);
    hipFree(mid_d);

    return mid_h;
}
