#include "hip/hip_runtime.h"
#include "imghelpercuda.h"
#include <vector>
#include <stdio.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hipfft/hipfft.h>

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void ImgHelperCuda::gpuAssert(hipError_t code, const char *file, int line, bool abort)
{
   if (code != hipSuccess)
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

void ImgHelperCuda::CheckCufftError(hipfftResult result, const char* method)
{
    switch(result)
    {
        case HIPFFT_SUCCESS:
            //printf("\n CUFFT (%s): SUCCESS", method);
            break;
        case HIPFFT_ALLOC_FAILED:
            printf("\n CUFFT (%s): Allocation failed", method);
            break;
        case HIPFFT_INVALID_VALUE:
            printf("\n CUFFT (%s): Invalid value", method);
            break;
        case HIPFFT_INTERNAL_ERROR:
            printf("\n CUFFT (%s): Internal error", method);
            break;
        case HIPFFT_SETUP_FAILED:
            printf("\n CUFFT (%s): Setup failed", method);
            break;
        case HIPFFT_INVALID_SIZE:
            printf("\n CUFFT (%s): Invalid size", method);
            break;
        default:
            printf("\n CUFFT (%s): unkown error", method);
    };
}

void ImgHelperCuda::fft(float* src, hipfftComplex* dst, int width, int height)
{
    float* plainSrc;
    hipfftComplex* plainDst;

    gpuErrchk(hipMalloc(&plainSrc, width * height * sizeof(float)));
    gpuErrchk(hipMalloc(&plainDst,  width * ((height/2) + 1) * sizeof(hipfftComplex)));
    gpuErrchk(hipMemcpy(plainSrc,src,width * height * sizeof(float),hipMemcpyHostToDevice));

    hipfftHandle handle;
    hipfftResult r = hipfftPlan2d(&handle,width,height,HIPFFT_R2C);
    CheckCufftError(r, "hipfftPlan2d");

    r = cufftSetCompatibilityMode(handle,CUFFT_COMPATIBILITY_NATIVE);
    CheckCufftError(r, "cufftSetCompatibilityMode");

    r = hipfftExecR2C(handle,plainSrc,plainDst);
    CheckCufftError(r, "hipfftExecR2C");
    hipDeviceSynchronize ();

    gpuErrchk(hipMemcpy(dst, plainDst, width * ((height/2) + 1) * sizeof(hipfftComplex),hipMemcpyDeviceToHost));

    r = hipfftDestroy(handle);
    CheckCufftError(r, "hipfftDestroy");

    hipFree(plainSrc);
    hipFree(plainDst);
}

hipfftComplex* ImgHelperCuda::fft2(float* src, int width, int height)
{
    float* plainSrc;
    hipfftComplex* plainDst;

    gpuErrchk(hipMalloc(&plainSrc, width * height * sizeof(float)));
    gpuErrchk(hipMalloc(&plainDst,  width * ((height/2) + 1) * sizeof(hipfftComplex)));
    gpuErrchk(hipMemcpy(plainSrc,src,width * height * sizeof(float),hipMemcpyHostToDevice));

    hipfftHandle handle;
    hipfftResult r = hipfftPlan2d(&handle,width,height,HIPFFT_R2C);
    CheckCufftError(r, "hipfftPlan2d");

    r = cufftSetCompatibilityMode(handle,CUFFT_COMPATIBILITY_NATIVE);
    CheckCufftError(r, "cufftSetCompatibilityMode");

    r = hipfftExecR2C(handle,plainSrc,plainDst);
    CheckCufftError(r, "hipfftExecR2C");
    hipDeviceSynchronize ();

    //gpuErrchk(hipMemcpy(dst, plainDst, width * ((height/2) + 1) * sizeof(hipfftComplex),hipMemcpyDeviceToHost));

    r = hipfftDestroy(handle);
    CheckCufftError(r, "hipfftDestroy");

    //hipFree(plainSrc);
    //hipFree(plainDst);
    return plainDst;
}

void ImgHelperCuda::ifft(hipfftComplex* src, float* dst, int width, int height)
{
    hipfftComplex* plainSrc;
    float* plainDst;

    gpuErrchk(hipMalloc(&plainSrc, width * ((height/2) + 1) * sizeof(hipfftComplex)));
    gpuErrchk(hipMalloc(&plainDst, width * height * sizeof(float)));
    gpuErrchk(hipMemcpy(plainSrc,src,width * ((height/2) + 1) * sizeof(hipfftComplex),hipMemcpyHostToDevice));

    hipfftHandle handle;
    hipfftResult r = hipfftPlan2d(&handle,width,height,HIPFFT_C2R);
    CheckCufftError(r, "hipfftPlan2d");

    r = cufftSetCompatibilityMode(handle,CUFFT_COMPATIBILITY_NATIVE);
    CheckCufftError(r, "cufftSetCompatibilityMode");

    r = hipfftExecC2R(handle,plainSrc,plainDst);
    CheckCufftError(r, "hipfftExecZ2D");
    hipDeviceSynchronize ();

    gpuErrchk(hipMemcpy(dst,plainDst,width * height * sizeof(float),hipMemcpyDeviceToHost));

    r = hipfftDestroy(handle);
    CheckCufftError(r, "hipfftDestroy");

    hipFree(plainSrc);
    hipFree(plainDst);
}

float* ImgHelperCuda::ifft2(hipfftComplex* src, int width, int height)
{
    //hipfftComplex* plainSrc;
    float* plainDst;

    //gpuErrchk(hipMalloc(&plainSrc, width * ((height/2) + 1) * sizeof(hipfftComplex)));
    gpuErrchk(hipMalloc(&plainDst, width * height * sizeof(float)));
    //gpuErrchk(hipMemcpy(plainSrc,src,width * ((height/2) + 1) * sizeof(hipfftComplex),hipMemcpyHostToDevice));

    hipfftHandle handle;
    hipfftResult r = hipfftPlan2d(&handle,width,height,HIPFFT_C2R);
    CheckCufftError(r, "hipfftPlan2d");

    r = cufftSetCompatibilityMode(handle,CUFFT_COMPATIBILITY_NATIVE);
    CheckCufftError(r, "cufftSetCompatibilityMode");

    r = hipfftExecC2R(handle,src,plainDst);
    CheckCufftError(r, "hipfftExecZ2D");
    hipDeviceSynchronize ();

    float* dst = (float*)malloc(width * height * sizeof(float));
    gpuErrchk(hipMemcpy(dst,plainDst,width * height * sizeof(float),hipMemcpyDeviceToHost));

    r = hipfftDestroy(handle);
    CheckCufftError(r, "hipfftDestroy");

    //hipFree(plainSrc);
    //hipFree(plainDst);

    return dst;
}


//this call could be avoided if we add this logic into the method ""ProcessNorm_intern"
__global__
void Process2DHT_intern(hipfftComplex* src, int gamma, int windowSize)
{
    int i = (blockIdx.x * blockDim.x) + threadIdx.x;
    int j = (blockIdx.y * blockDim.y) + threadIdx.y;
    int pos = i * windowSize + j;
    if(pos < windowSize * windowSize)
    {
        //avoid if (perf)!!! - GPU branching!!!
        if(src[pos].x < 0 && (src[pos].x * -1) < gamma ) { src[pos].x = 0; src[pos].y = 0; }
        if(src[pos].x > 0 && src[pos].x < gamma) { src[pos].x = 0; src[pos].y = 0; }
    }
}

__global__
void ProcessNorm_intern(hipfftComplex* src, float* normVector, int windowSize, int blockSize)
{
    int i = (blockIdx.x * blockDim.x) + threadIdx.x;
    int j = (blockIdx.y * blockDim.y) + threadIdx.y;
    int pos = i * windowSize + j;
    if((pos < (windowSize * windowSize)) && (pos % blockSize == 0))
    {
        int outIndex = (pos /blockSize) -1;
        //perf O(n2)!!!!
        float norm = 0;
        for(int k =0; i < blockSize; ++i)
        {
            for(int n=0; k < blockSize; ++k)
            {
                int i2 = i + k;
                int j2 = j + n;
                int pos2 = i2 * windowSize + j2;

                //Verify the formula!!!
                if(pos2 < (windowSize * windowSize)) norm += (src[pos2].x * src[pos2].x);
            }
        }
        normVector[outIndex] = norm;
    }
}

hipfftComplex* ImgHelperCuda::get(hipfftComplex* src, int width, int height)
{
    hipfftComplex* dst = (hipfftComplex*)malloc(width * (height/2) * sizeof(hipfftComplex));
    gpuErrchk(hipMemcpy(dst,src,width * height * sizeof(float),hipMemcpyDeviceToHost));
    return dst;
}

float* ImgHelperCuda::get(float* src, int width, int height)
{
    float* dst = (float*)malloc(width * height * sizeof(float));
    gpuErrchk(hipMemcpy(dst,src,width * height * sizeof(float),hipMemcpyDeviceToHost));
    return dst;
}

void ImgHelperCuda::ProcessBM(hipfftComplex* src, int gamma, int windowSize, int blockSize)
{
    dim3 threadsPerBlock(ImgHelperCuda::HT_2D_THREADS, ImgHelperCuda::HT_2D_THREADS);
    dim3 numBlocks(windowSize/threadsPerBlock.x, windowSize/threadsPerBlock.y);

    printf("\n\tprocess 2D HT");
    Process2DHT_intern<<<numBlocks,threadsPerBlock>>>(src, gamma, windowSize);
    hipDeviceSynchronize();

    printf("\n\tComputes blocks value");
    float* normVector_d;
    int sizeNormVector = (windowSize / blockSize) ;
    gpuErrchk(hipMalloc(&normVector_d, sizeNormVector * sizeNormVector * sizeof(float)));

    ///--> we have to reduce the number of blocks and threads!!!!
    ProcessNorm_intern<<<numBlocks,threadsPerBlock>>>(src, normVector_d, windowSize, blockSize);
    hipDeviceSynchronize();

    //Only for testing
    float* normVector_h = (float*)malloc(sizeNormVector * sizeNormVector * sizeof(float));
    gpuErrchk(hipMemcpy(normVector_h,normVector_d, sizeNormVector * sizeNormVector * sizeof(float), hipMemcpyDeviceToHost));
    printf("\n\n----- BLOCKS VALUE (TEST) ------\n");
    for(int i= 0; i < sizeNormVector * sizeNormVector; ++i)
    {
        printf("B%i: %f\n", (i+1), normVector_h[i] );
    }
    printf("\n");

    printf("\n\tMatching - 3D groups");
    printf("\n");
}




