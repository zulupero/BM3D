#include "hip/hip_runtime.h"
#include <stdio.h>

#include "bm3d.h"
#include "utilities.h"
#include "timeutil.h"

#include <string>

BM3D::BM3D_Context BM3D::context;

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void BM3D::gpuAssert(hipError_t code, const char *file, int line, bool abort)
{
   if (code != hipSuccess)
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}


void BM3D::BM3D_dispose()
{
}

void BM3D::BM3D_Initialize(BM3D::SourceImage img, int width, int height, int pHard, int hardLimit, double hardThreshold, int sigma, bool debug)
{
    printf("\n--> Execution on Tesla K40c");
    if(hipSuccess != hipSetDevice(0)) printf("\n\tNo device 0 available");

    if(debug)
    {
        int sz = 1048576 * 1024;
        hipDeviceSetLimit(hipLimitPrintfFifoSize, sz);
    }

    printf("\nBM3D context initialization");
    BM3D::context.debugPixel = 1792;
    BM3D::context.debugBlock = 15508;

    BM3D::context.hardLimit = hardLimit;
    BM3D::context.hardThreshold = hardThreshold;
    BM3D::context.sigma = sigma;

    int w2 = width - (width % pHard) + (10 * pHard);
    int h2 = height - (height % pHard) + (10 * pHard);
    
    BM3D::context.nbBlocksIntern = (width / pHard) * (height /pHard);
    BM3D::context.nbBlocks = (w2 / pHard) * (h2 /pHard);
    BM3D::context.widthBlocks = (w2 / pHard);
    BM3D::context.widthBlocksIntern = (width / pHard);

    w2 += 8;  //nHard = 8
    h2 += 8;  //nHard = 8
    
    BM3D::context.img_widthOrig = width; 
    BM3D::context.img_heightOrig= height;
    BM3D::context.img_width = w2; 
    BM3D::context.img_height= h2;
    BM3D::context.pHard = pHard;
    BM3D::context.sourceImage = img;

    gpuErrchk(hipMalloc(&BM3D::context.deviceImage, width * height * sizeof(float)));
    gpuErrchk(hipMemcpy(BM3D::context.deviceImage, &img[0], width * height * sizeof(float), hipMemcpyHostToDevice));
    gpuErrchk(hipMalloc(&BM3D::context.basicImage, w2 * h2 * sizeof(float)));
    gpuErrchk(hipMemset(BM3D::context.basicImage, 0, w2 * h2 * sizeof(float)));
    gpuErrchk(hipMalloc(&BM3D::context.estimates, w2 * h2 * 2 * sizeof(float)));
    gpuErrchk(hipMemset(BM3D::context.estimates, 0, w2 * h2 * 2 * sizeof(float)));
    //Kaiser-window coef
    float kaiserWindow[64] = {  0.1924, 0.2989, 0.3846, 0.4325, 0.4325, 0.3846, 0.2989, 0.1924,
                                0.2989, 0.4642, 0.5974, 0.6717, 0.6717, 0.5974, 0.4642, 0.2989,
                                0.3846, 0.5974, 0.7688, 0.8644, 0.8644, 0.7688, 0.5974, 0.3846,
                                0.4325, 0.6717, 0.8644, 0.9718, 0.9718, 0.8644, 0.6717, 0.4325,
                                0.4325, 0.6717, 0.8644, 0.9718, 0.9718, 0.8644, 0.6717, 0.4325,
                                0.3846, 0.5974, 0.7688, 0.8644, 0.8644, 0.7688, 0.5974, 0.3846,
                                0.2989, 0.4642, 0.5974, 0.6717, 0.6717, 0.5974, 0.4642, 0.2989,
                                0.1924, 0.2989, 0.3846, 0.4325, 0.4325, 0.3846, 0.2989, 0.1924};

    gpuErrchk(hipMalloc(&BM3D::context.kaiserWindowCoef, 64 * sizeof(float)));
    gpuErrchk(hipMemcpy(BM3D::context.kaiserWindowCoef, kaiserWindow, 64 * sizeof(float), hipMemcpyHostToDevice));
    gpuErrchk(hipMalloc(&BM3D::context.blockMap, BM3D::context.nbBlocksIntern * 100 * 10 * sizeof(int)));
    gpuErrchk(hipMalloc(&BM3D::context.blocks, BM3D::context.nbBlocks * 66 * sizeof(double)));
    gpuErrchk(hipMalloc(&BM3D::context.blocksOrig, BM3D::context.nbBlocks * 66 * sizeof(double)));
    gpuErrchk(hipMalloc(&BM3D::context.bmVectors, BM3D::context.nbBlocksIntern * 16 * sizeof(int)));
    gpuErrchk(hipMemset(BM3D::context.bmVectors, 0, BM3D::context.nbBlocksIntern * 16 * sizeof(int)));
    gpuErrchk(hipMalloc(&BM3D::context.blocks3D, BM3D::context.nbBlocksIntern * 16 * 64 * sizeof(double)));
    gpuErrchk(hipMalloc(&BM3D::context.blocks3DOrig, BM3D::context.nbBlocksIntern * 16 * 64 * sizeof(double)));
    gpuErrchk(hipMalloc(&BM3D::context.npArray, BM3D::context.nbBlocksIntern  * sizeof(int)));
    gpuErrchk(hipMemset(BM3D::context.npArray, 0, BM3D::context.nbBlocksIntern  * sizeof(int)));
    gpuErrchk(hipMalloc(&BM3D::context.wpArray, BM3D::context.nbBlocksIntern  * sizeof(double)));
    gpuErrchk(hipMalloc(&BM3D::context.nbSimilarBlocks, BM3D::context.nbBlocksIntern  * sizeof(float)));
    gpuErrchk(hipMemset(BM3D::context.nbSimilarBlocks, 0, BM3D::context.nbBlocksIntern  * sizeof(float)));

    printf("\n\tNumber of blocks          = %d", BM3D::context.nbBlocks);
    printf("\n\tNumber of blocks (intern) = %d", BM3D::context.nbBlocksIntern);
    printf("\n\tWidth blocks (intern)     = %d", BM3D::context.widthBlocksIntern);
    printf("\n\tWidth blocks              = %d", BM3D::context.widthBlocks);
    printf("\n\tWidth                     = %d", BM3D::context.img_width);
    printf("\n\tHeight                    = %d", BM3D::context.img_height);
    printf("\n\tDevice image              = %f Mb", (width * height * sizeof(float)/1024.00 / 1024.00));  
    printf("\n\tBasic image               = %f Mb", (w2 * h2 * sizeof(float)/1024.00 / 1024.00));
    printf("\n\tBlocks array              = %f Mb", (BM3D::context.nbBlocks * 66 * sizeof(double)/1024.00 / 1024.00));  
    printf("\n\tBlocks array (orig)       = %f Mb", (BM3D::context.nbBlocks * 66 * sizeof(double)/1024.00 / 1024.00));  
    printf("\n\tBlocks map                = %f Mb", (BM3D::context.nbBlocks * 100 * 10 * sizeof(int)/1024.00 / 1024.00));  
    printf("\n\tBM vectors                = %f Mb", (BM3D::context.nbBlocksIntern * 16 * sizeof(int)/1024.00 / 1024.00)); 
    printf("\n\tBlocks 3D                 = %f Mb", (BM3D::context.nbBlocksIntern * 16 * 64 * sizeof(double)/1024.00 / 1024.00));  
    printf("\n\tBlocks 3D (orig)          = %f Mb", (BM3D::context.nbBlocksIntern * 16 * 64 * sizeof(double)/1024.00 / 1024.00));
    printf("\n\tNP array                  = %f Mb", (BM3D::context.nbBlocksIntern  * sizeof(int)/1024.00 / 1024.00));
    printf("\n\tWP array                  = %f Mb", (BM3D::context.nbBlocksIntern  * sizeof(double)/1024.00 / 1024.00));
    printf("\n\tEstimates array           = %f Mb", (w2 * h2 * 2 * sizeof(float)/1024.00 / 1024.00));
    printf("\n\tSimilar blocks array      = %f Mb", (BM3D::context.nbBlocksIntern  * sizeof(float)/1024.00 / 1024.00));

}

void BM3D::BM3D_Run()
{
    printf("\n\nRun BM3D"); 
    BM3D_BasicEstimate();
    BM3D_FinalEstimate();   
}

void BM3D::BM3D_SaveBasicImage()
{
    float* basicImage = (float*)malloc(BM3D::context.img_widthOrig * BM3D::context.img_heightOrig * sizeof(float));
    gpuErrchk(hipMemcpy(&basicImage[0], BM3D::context.deviceImage, BM3D::context.img_widthOrig * BM3D::context.img_heightOrig * sizeof(float), hipMemcpyDeviceToHost));
    std::string filename("test.png");
    save_image(filename.c_str(), basicImage, BM3D::context.img_widthOrig, BM3D::context.img_heightOrig, 1);
}

void BM3D::BM3D_FinalEstimate()
{
    printf("\n\tFinal estimates (2 step)");
    Timer::start(); 
    BM3D_CreateBlock();
    BM3D_2DTransform(true);
    BM3D_BlockMatching(true);
    BM3D_WienFilter();
    Timer::add("BM3D-Final estimates");
}

void BM3D::BM3D_BasicEstimate()
{
    printf("\n\tBasic estimates (1 step)");
    Timer::start();
    BM3D_CreateBlock();
    BM3D_2DTransform();
    BM3D_BlockMatching();
    BM3D_HardThresholdFilter();
    BM3D_Inverse3D();
    BM3D_Aggregation();
    BM3D_InverseShift();
    Timer::add("BM3D-Basic estimates");
    BM3D_SaveBasicImage();
}

void WienFilter()
{
}

void BM3D::BM3D_WienFilter()
{
    {
        /*dim3 numBlocks(BM3D::context.widthBlocksIntern, BM3D::context.widthBlocksIntern, 16);
        dim3 numThreads(8, 8);
        WienFilter<<<numBlocks,numThreads>>>(); 
        hipDeviceSynchronize();   */
    }
}

__global__
void pre_aggregation(double* blocks3D, double* wpArray, double* blocks, int* bmVectors, int size, float* kaiserCoef, float* estimates, int width)
{
    int block = (blockIdx.y * size) + blockIdx.x;
    int bmVectorIndex = (block << 4) + blockIdx.z;
    int blockIndex = bmVectors[bmVectorIndex];
    if(blockIndex > 0)
    {
        int xImg = (int)blocks[blockIndex + 64];
        int yImg = (int)blocks[blockIndex + 65];
        int xPixel = xImg + threadIdx.x;
        int yPixel = yImg + threadIdx.y;
        int estimateIndex = ((yPixel * width) + xPixel) << 1;
        int kaiserIndex = (threadIdx.y << 3) + threadIdx.x;
        int block3DIndex = (block << 10) + (blockIdx.z << 6) + kaiserIndex;
        atomicAdd(&estimates[estimateIndex], (kaiserCoef[kaiserIndex] * wpArray[block] * blocks3D[block3DIndex]));
        atomicAdd(&estimates[estimateIndex+1], (kaiserCoef[kaiserIndex] * wpArray[block]));
    }
}

__global__
void aggregation(float* estimates, float* basicImage, int size)
{
    int basicImageIndex = (((blockIdx.y * size) + blockIdx.x) << 6) + (threadIdx.y << 3) + threadIdx.x;
    int estimateIndex = (basicImageIndex << 1);
    basicImage[basicImageIndex] = estimates[estimateIndex]/estimates[estimateIndex+1];
}

void BM3D::BM3D_Aggregation()
{
    {
        dim3 numBlocks(BM3D::context.widthBlocksIntern, BM3D::context.widthBlocksIntern, 16);
        dim3 numThreads(8, 8);
        pre_aggregation<<<numBlocks,numThreads>>>(BM3D::context.blocks3D, BM3D::context.wpArray, BM3D::context.blocks, BM3D::context.bmVectors, BM3D::context.widthBlocksIntern, BM3D::context.kaiserWindowCoef, BM3D::context.estimates, BM3D::context.img_width); 
        hipDeviceSynchronize();   
    }
    {
        dim3 numBlocks(BM3D::context.img_width/8, BM3D::context.img_height/8);
        dim3 numThreads(8, 8);
        aggregation<<<numBlocks,numThreads>>>(BM3D::context.estimates, BM3D::context.basicImage, BM3D::context.img_height/8); 
        hipDeviceSynchronize();   
    }
}

__global__
void inverse3D(double* blocks3D, int size)
{
    int block3DIndex = (((blockIdx.y * size) + blockIdx.x) << 10) + (threadIdx.y << 3) + threadIdx.x;

    double a = blocks3D[block3DIndex];
    double b = blocks3D[block3DIndex+64];
    double c = blocks3D[block3DIndex+128];
    double d = blocks3D[block3DIndex+192];
    double e = blocks3D[block3DIndex+256];
    double f = blocks3D[block3DIndex+320];
    double g = blocks3D[block3DIndex+384];
    double h = blocks3D[block3DIndex+448];
    double i = blocks3D[block3DIndex+512];
    double j = blocks3D[block3DIndex+576];
    double k = blocks3D[block3DIndex+640];
    double l = blocks3D[block3DIndex+704];
    double m = blocks3D[block3DIndex+768];
    double n = blocks3D[block3DIndex+832];
    double o = blocks3D[block3DIndex+896];
    double p = blocks3D[block3DIndex+960];

    blocks3D[block3DIndex] = (a+b+c+d+e+f+g+h+i+j+k+l+m+n+o+p) / 4.0;
    blocks3D[block3DIndex+64] = (a-b+c-d+e-f+g-h+i-j+k-l+m-n+o-p) / 4.0;
    blocks3D[block3DIndex+128] = (a+b-c-d+e+f-g-h+i+j-k-l+m+n-o-p) / 4.0;
    blocks3D[block3DIndex+192] = (a-b-c+d+e-f-g+h+i-j-k+l+m-n-o+p) / 4.0;
    blocks3D[block3DIndex+256] = (a+b+c+d-e-f-g-h+i+j+k+l-m-n-o-p) / 4.0;
    blocks3D[block3DIndex+320] = (a-b+c-d-e+f-g+h+i-j+k-l-m+n-o+p) / 4.0;
    blocks3D[block3DIndex+384] = (a+b-c-d-e-f+g+h+i+j-k-l-m-n+o+p) / 4.0;
    blocks3D[block3DIndex+448] = (a-b-c+d-e+f+g-h+i-j-k+l-m+n+o-p) / 4.0;
    blocks3D[block3DIndex+512] = (a+b+c+d+e+f+g+h-i-j-k-l-m-n-o-p) / 4.0;
    blocks3D[block3DIndex+576] = (a-b+c-d+e-f+g-h-i+j-k+l-m+n-o+p) / 4.0;
    blocks3D[block3DIndex+640] = (a+b-c-d+e+f-g-h-i-j+k+l-m-n+o+p) / 4.0;
    blocks3D[block3DIndex+704] = (a-b-c+d+e-f-g+h-i+j+k-l-m+n+o-p) / 4.0;
    blocks3D[block3DIndex+768] = (a+b+c+d-e-f-g-h-i-j-k-l+m+n+o+p) / 4.0;
    blocks3D[block3DIndex+832] = (a-b+c-d-e+f-g+h-i+j-k+l+m-n+o-p) / 4.0;
    blocks3D[block3DIndex+896] = (a+b-c-d-e-f+g+h-i-j+k+l+m+n-o-p) / 4.0;
    blocks3D[block3DIndex+960] = (a-b-c+d-e+f+g-h-i+j+k-l+m-n-o+p) / 4.0;
}


__device__ void Hadamar8(double* inputs, double DIVISOR)
{
    double a = inputs[0];
    double b = inputs[1];
    double c = inputs[2];
    double d = inputs[3];
    double e = inputs[4];  
    double f = inputs[5];
    double g = inputs[6];
    double h = inputs[7];
    
    inputs[0] = (a+b+c+d+e+f+g+h)/DIVISOR;
    inputs[1] = (a-b+c-d+e-f+g-h)/DIVISOR;
    inputs[2] = (a+b-c-d+e+f-g-h)/DIVISOR;
    inputs[3] = (a-b-c+d+e-f-g+h)/DIVISOR;
    inputs[4] = (a+b+c+d-e-f-g-h)/DIVISOR;
    inputs[5] = (a-b+c-d-e+f-g+h)/DIVISOR;
    inputs[6] = (a+b-c-d-e-f+g+h)/DIVISOR;
    inputs[7] = (a-b-c+d-e+f+g-h)/DIVISOR;
}

__global__
void inverseTransform2D_row(double* blocks3D, int size, double DIVISOR)
{
    int blockIndex = (((blockIdx.y * size) + blockIdx.x) << 10) + (threadIdx.y << 6) + (threadIdx.x << 3);
    double inputs[8];
    inputs[0] = blocks3D[blockIndex];
    inputs[1] = blocks3D[blockIndex+1];
    inputs[2] = blocks3D[blockIndex+2];
    inputs[3] = blocks3D[blockIndex+3];
    inputs[4] = blocks3D[blockIndex+4];
    inputs[5] = blocks3D[blockIndex+5];
    inputs[6] = blocks3D[blockIndex+6];
    inputs[7] = blocks3D[blockIndex+7];
    Hadamar8(inputs, DIVISOR);
    blocks3D[blockIndex] = inputs[0];
    blocks3D[blockIndex+1] = inputs[1];
    blocks3D[blockIndex+2] = inputs[2];
    blocks3D[blockIndex+3] = inputs[3];
    blocks3D[blockIndex+4] = inputs[4];
    blocks3D[blockIndex+5] = inputs[5];
    blocks3D[blockIndex+6] = inputs[6];
    blocks3D[blockIndex+7] = inputs[7];
}

__global__
void inverseTransform2D_col(double* blocks3D, int size, double DIVISOR)
{
    int blockIndex = (((blockIdx.y * size) + blockIdx.x) << 10) + (threadIdx.y << 6) + threadIdx.x;
    double inputs[8];
    int index = blockIndex;
    inputs[0] = blocks3D[index];
    index += 8;
    inputs[1] = blocks3D[index];
    index += 8;
    inputs[2] = blocks3D[index];
    index += 8;
    inputs[3] = blocks3D[index];
    index += 8;
    inputs[4] = blocks3D[index];
    index += 8;
    inputs[5] = blocks3D[index];
    index += 8;
    inputs[6] = blocks3D[index];
    index += 8;
    inputs[7] = blocks3D[index];
    Hadamar8(inputs, DIVISOR);
    index = blockIndex;
    blocks3D[index] = inputs[0];
    index += 8;
    blocks3D[index] = inputs[1];
    index += 8;
    blocks3D[index] = inputs[2];
    index += 8;
    blocks3D[index] = inputs[3];
    index += 8;
    blocks3D[index] = inputs[4];
    index += 8;
    blocks3D[index] = inputs[5];
    index += 8;
    blocks3D[index] = inputs[6];
    index += 8;
    blocks3D[index] = inputs[7];
}


void BM3D::BM3D_Inverse3D()
{
    double DIVISOR = sqrt(8);
    {
        dim3 numBlocks(BM3D::context.widthBlocksIntern, BM3D::context.widthBlocksIntern);
        dim3 numThreads(8, 8);
        inverse3D<<<numBlocks,numThreads>>>(BM3D::context.blocks3D, BM3D::context.widthBlocksIntern); 
        hipDeviceSynchronize();   
    }
    {
        dim3 numBlocks(BM3D::context.widthBlocksIntern, BM3D::context.widthBlocksIntern);
        dim3 numThreads(8, 16);
        inverseTransform2D_col<<<numBlocks,numThreads>>>(BM3D::context.blocks3D, BM3D::context.widthBlocksIntern, DIVISOR); 
        hipDeviceSynchronize();   
    }
    {
        dim3 numBlocks(BM3D::context.widthBlocksIntern, BM3D::context.widthBlocksIntern);
        dim3 numThreads(8, 16);
        inverseTransform2D_row<<<numBlocks,numThreads>>>(BM3D::context.blocks3D, BM3D::context.widthBlocksIntern, DIVISOR); 
        hipDeviceSynchronize();   
    }
}

__global__
void HardThresholdFilter(double* blocks3D, double threshold, int size, float* nbSimilarBlocks)
{
    int block = (blockIdx.y * size) + blockIdx.x;
    int blockPixelIndex = (block << 10) + (blockIdx.z << 6) + (threadIdx.y << 3) + threadIdx.x;
    if(blocks3D[blockPixelIndex] < (threshold /** nbSimilarBlocks[block]*/)) blocks3D[blockPixelIndex] = 0;  
}

__global__
void CalculateNP(double* blocks3D, int* npArray, int size)
{
    int block = ((blockIdx.y * size) + blockIdx.x);
    int blockIndex = (block << 10) + (threadIdx.y << 3) + (threadIdx.y << 5) + threadIdx.x;
    if(blocks3D[blockIndex] > 0) atomicAdd(&npArray[block], 1);  
}


__global__
void CalculateWP(double* wpArray, int* npArray, int size, int sigma)
{
    int block = ((blockIdx.y * size) + blockIdx.x);
    wpArray[block] = (npArray[block] > 1) ? (1.0 / (sigma * sigma * npArray[block])) : 1.0;
}


void BM3D::BM3D_HardThresholdFilter()
{
    {
        dim3 numBlocks(BM3D::context.widthBlocksIntern, BM3D::context.widthBlocksIntern, 16);
        dim3 numThreads(8, 8);
        HardThresholdFilter<<<numBlocks,numThreads>>>(BM3D::context.blocks3D, BM3D::context.hardThreshold, BM3D::context.widthBlocksIntern, BM3D::context.nbSimilarBlocks); 
        hipDeviceSynchronize();   
    }
    {
        dim3 numBlocks(BM3D::context.widthBlocksIntern, BM3D::context.widthBlocksIntern);
        dim3 numThreads(32, 32);
        CalculateNP<<<numBlocks,numThreads>>>(BM3D::context.blocks3D, BM3D::context.npArray, BM3D::context.widthBlocksIntern); 
        hipDeviceSynchronize();   
    }
    {
        dim3 numBlocks(BM3D::context.widthBlocksIntern, BM3D::context.widthBlocksIntern);
        dim3 numThreads(1);
        CalculateWP<<<numBlocks,numThreads>>>(BM3D::context.wpArray, BM3D::context.npArray, BM3D::context.widthBlocksIntern, BM3D::context.sigma); 
        hipDeviceSynchronize();   
    }
}

__global__
void showDistance(int x, int y, int size, int* blockMap, double* blocks, int* bmVectors)
{
    //int index = ((y * size) + x) * 1000;
    int bmVectorIndex = ((y * size) + x) << 4;
    printf("\n");
    printf("\nBM Vector = ");
    for(int i=0; i<16; ++i) printf(" %d, ", (bmVectors[bmVectorIndex + i] / 66));
    /*for(int i= 0; i < 100; i++)
    {
        //if(blockMap[index + (i * 10) + 9] < 99999999)
        {
            int blockIndex = blockMap[index + (i* 10)];
            printf("\ncmp block %d", i);
            printf("\nindex: %d", blockIndex);
            printf("\nblock x=%d, y=%d", (int)blocks[blockIndex+64], (int)blocks[blockIndex+65]);
            for(int j = 0; j< 9; ++j)
                printf("\n\t%d: distance %d", j, blockMap[index + (i * 10) + 1 + j]);
        }
    }*/
}

void BM3D::BM3D_ShowDistance(int x, int y)
{
   {
        dim3 numBlocks(1);
        dim3 numThreads(1);
        showDistance<<<numBlocks,numThreads>>>(x, y, BM3D::context.widthBlocksIntern, BM3D::context.blockMap, BM3D::context.blocks, BM3D::context.bmVectors); 
        hipDeviceSynchronize();   
   }     
}

__global__
void ShowPixel(int x, int y, int nbBlocks, double* blocks, float* deviceImage, float* basicImage, int widthOrig, int width)
{
    for(int i=0; i < nbBlocks; ++i)
    {
        float xImg = blocks[(i * 66) + 64];
        float yImg = blocks[(i * 66) + 65];
        if(x >= xImg && x <= (xImg + 8) && y >= yImg && y <= (yImg + 8))
            printf("\nblock = %d, x = %d, y = %d, xImg = %f, yImg = %f", i, x, y, xImg, yImg );
    }
    
    int origImageIndex = (y * widthOrig) +x;
    int basicImageIndex = ((y+15) * width) +(x+15);
    printf("\nOrig value = %f", deviceImage[origImageIndex]);
    printf("\nestimate value = %f", basicImage[basicImageIndex]);
        
}

void BM3D::BM3D_ShowPixel(int x, int y)
{
   {
        dim3 numBlocks(1);
        dim3 numThreads(1);
        ShowPixel<<<numBlocks,numThreads>>>(x, y, BM3D::context.nbBlocks, BM3D::context.blocks, BM3D::context.deviceImage, BM3D::context.basicImage,  BM3D::context.img_widthOrig, BM3D::context.img_width); 
        hipDeviceSynchronize();   
   }     
}

__global__
void BM_CalculateDistance(int* blockMap, double* blocks, int size)
{
    int blockMapIndex = (((blockIdx.y * size) + blockIdx.x) * 1000) + ((threadIdx.y * 10 + threadIdx.x) * 10);
    int cmpBlockMapIndex = (((blockIdx.y * size) + blockIdx.x) * 1000) + 550;
    int cmpBlockIndex = blockMap[cmpBlockMapIndex] + (threadIdx.z << 3);
    int blockIndex =  blockMap[blockMapIndex] + (threadIdx.z << 3);
      
    blockMap[blockMapIndex + 1 + threadIdx.z] = int(
          
                                ((blocks[cmpBlockIndex] - blocks[blockIndex]) * (blocks[cmpBlockIndex] - blocks[blockIndex])) +
                                ((blocks[cmpBlockIndex+1] - blocks[blockIndex+1]) * (blocks[cmpBlockIndex+1] - blocks[blockIndex+1])) +
                                ((blocks[cmpBlockIndex+2] - blocks[blockIndex+2]) * (blocks[cmpBlockIndex+2] - blocks[blockIndex+2])) +
                                ((blocks[cmpBlockIndex+3] - blocks[blockIndex+3]) * (blocks[cmpBlockIndex+3] - blocks[blockIndex+3])) +
                                ((blocks[cmpBlockIndex+4] - blocks[blockIndex+4]) * (blocks[cmpBlockIndex+4] - blocks[blockIndex+4])) +
                                ((blocks[cmpBlockIndex+5] - blocks[blockIndex+5]) * (blocks[cmpBlockIndex+5] - blocks[blockIndex+5])) +
                                ((blocks[cmpBlockIndex+6] - blocks[blockIndex+6]) * (blocks[cmpBlockIndex+6] - blocks[blockIndex+6])) +
                                ((blocks[cmpBlockIndex+7] - blocks[blockIndex+7]) * (blocks[cmpBlockIndex+7] - blocks[blockIndex+7])));
}

__global__
void BM_AddAndLimit(int* blockMap, int size, int limit)
{
    int blockMapIndex = (((blockIdx.y * size) + blockIdx.x) * 1000) + (threadIdx.y * 10 + threadIdx.x) * 10;
    int sum = blockMap[blockMapIndex + 1] + blockMap[blockMapIndex + 2] + blockMap[blockMapIndex + 3] + blockMap[blockMapIndex + 4] + blockMap[blockMapIndex + 5] + blockMap[blockMapIndex + 6] + blockMap[blockMapIndex + 7] + blockMap[blockMapIndex + 8];
    blockMap[blockMapIndex + 9] = (sum <= limit) ? sum : 99999999;
}


__global__
void BM_Sort(int* blockMap, int size)
{
    int blockMapIndex = (((blockIdx.y * size) + blockIdx.x) * 1000);
    int currentBlockIndex = blockMapIndex + (threadIdx.y * 100) + (threadIdx.x *10);
    int currentD = blockMap[currentBlockIndex+9];
    if(currentD < 99999999)
    {
        int index = 0;
        for(int i=0; i<100; ++i)
        {
            if(currentD > blockMap[blockMapIndex + i * 10 + 9]) index++;
        }
        blockMap[currentBlockIndex+1] = index;
    }
}

__global__
void BM_CreateBmVector(int* blockMap, int* bmVectors, int size, float* nbSimilarBlocks)
{
    int currentBlockIndex = (((blockIdx.y * size) + blockIdx.x) * 1000) + (threadIdx.y * 100) + (threadIdx.x *10);
    int bmVectorIndex = (((blockIdx.y * size) + blockIdx.x) << 4);
    if(blockMap[currentBlockIndex+9] < 99999999)
    {
        bmVectors[bmVectorIndex + blockMap[currentBlockIndex+1]] = blockMap[currentBlockIndex];
        atomicAdd(&nbSimilarBlocks[(blockIdx.y * size) + blockIdx.x], 1);
    }
}

__global__
void Create3DBlocks(double* blocks, double* blocks3D, int* bmVectors, int size)
{
    int index = (blockIdx.y * size) + blockIdx.x;
    int bmVectorIndex = index << 4;
    {
        
        int pixelIndex = (threadIdx.y << 3) + threadIdx.x;
        int block3DIndex = (index << 10) + pixelIndex;
        //we can assume that the top-left corner of the basic image always has a pixel egal to 0 due to the shift 
        //of the image. 

        double a = blocks[bmVectors[bmVectorIndex] + pixelIndex];
        double b = blocks[bmVectors[bmVectorIndex+1] + pixelIndex];
        double c = blocks[bmVectors[bmVectorIndex+2] + pixelIndex];
        double d = blocks[bmVectors[bmVectorIndex+3] + pixelIndex];
        double e = blocks[bmVectors[bmVectorIndex+4] + pixelIndex];
        double f = blocks[bmVectors[bmVectorIndex+5] + pixelIndex];
        double g = blocks[bmVectors[bmVectorIndex+6] + pixelIndex];
        double h = blocks[bmVectors[bmVectorIndex+7] + pixelIndex];
        double i = blocks[bmVectors[bmVectorIndex+8] + pixelIndex];
        double j = blocks[bmVectors[bmVectorIndex+9] + pixelIndex];
        double k = blocks[bmVectors[bmVectorIndex+10] + pixelIndex];
        double l = blocks[bmVectors[bmVectorIndex+11] + pixelIndex];
        double m = blocks[bmVectors[bmVectorIndex+12] + pixelIndex];
        double n = blocks[bmVectors[bmVectorIndex+13] + pixelIndex];
        double o = blocks[bmVectors[bmVectorIndex+14] + pixelIndex];
        double p = blocks[bmVectors[bmVectorIndex+15] + pixelIndex];

        
        blocks3D[block3DIndex] = (a+b+c+d+e+f+g+h+i+j+k+l+m+n+o+p) / 4.0;
        blocks3D[block3DIndex+64] = (a-b+c-d+e-f+g-h+i-j+k-l+m-n+o-p) / 4.0;
        blocks3D[block3DIndex+128] = (a+b-c-d+e+f-g-h+i+j-k-l+m+n-o-p) / 4.0;
        blocks3D[block3DIndex+192] = (a-b-c+d+e-f-g+h+i-j-k+l+m-n-o+p) / 4.0;
        blocks3D[block3DIndex+256] = (a+b+c+d-e-f-g-h+i+j+k+l-m-n-o-p) / 4.0;
        blocks3D[block3DIndex+320] = (a-b+c-d-e+f-g+h+i-j+k-l-m+n-o+p) / 4.0;
        blocks3D[block3DIndex+384] = (a+b-c-d-e-f+g+h+i+j-k-l-m-n+o+p) / 4.0;
        blocks3D[block3DIndex+448] = (a-b-c+d-e+f+g-h+i-j-k+l-m+n+o-p) / 4.0;
        blocks3D[block3DIndex+512] = (a+b+c+d+e+f+g+h-i-j-k-l-m-n-o-p) / 4.0;
        blocks3D[block3DIndex+576] = (a-b+c-d+e-f+g-h-i+j-k+l-m+n-o+p) / 4.0;
        blocks3D[block3DIndex+640] = (a+b-c-d+e+f-g-h-i-j+k+l-m-n+o+p) / 4.0;
        blocks3D[block3DIndex+704] = (a-b-c+d+e-f-g+h-i+j+k-l-m+n+o-p) / 4.0;
        blocks3D[block3DIndex+768] = (a+b+c+d-e-f-g-h-i-j-k-l+m+n+o+p) / 4.0;
        blocks3D[block3DIndex+832] = (a-b+c-d-e+f-g+h-i+j-k+l+m-n+o-p) / 4.0;
        blocks3D[block3DIndex+896] = (a+b-c-d-e-f+g+h-i-j+k+l+m+n-o-p) / 4.0;
        blocks3D[block3DIndex+960] = (a-b-c+d-e+f+g-h-i+j+k-l+m-n-o+p) / 4.0;

    }
}

__global__
void Create3DBlocksOrig(double* blocks, double* blocks3DOrig, float* noisyImage, int* bmVectors, int size)
{
}

__global__
void CalculateCoefSimilarBlocks(float* similarBlocks, int size)
{
    similarBlocks[(blockIdx.y * size) + blockIdx.x] = sqrtf(similarBlocks[(blockIdx.y * size) + blockIdx.x]);
}

void BM3D::BM3D_BlockMatching(bool final)
{
    {
        dim3 numBlocks(BM3D::context.widthBlocksIntern, BM3D::context.widthBlocksIntern);
        dim3 numThreads(10, 10, 8);
        BM_CalculateDistance<<<numBlocks,numThreads>>>(BM3D::context.blockMap, BM3D::context.blocks, BM3D::context.widthBlocksIntern); 
        hipDeviceSynchronize();   
    }
    {
        dim3 numBlocks(BM3D::context.widthBlocksIntern, BM3D::context.widthBlocksIntern);
        dim3 numThreads(10, 10);
        BM_AddAndLimit<<<numBlocks,numThreads>>>(BM3D::context.blockMap, BM3D::context.widthBlocksIntern, BM3D::context.hardLimit); 
        hipDeviceSynchronize();   
    }
    {
        dim3 numBlocks(BM3D::context.widthBlocksIntern, BM3D::context.widthBlocksIntern);
        dim3 numThreads(10, 10);
        BM_Sort<<<numBlocks,numThreads>>>(BM3D::context.blockMap, BM3D::context.widthBlocksIntern); 
        hipDeviceSynchronize();    
        BM_CreateBmVector<<<numBlocks,numThreads>>>(BM3D::context.blockMap, BM3D::context.bmVectors, BM3D::context.widthBlocksIntern, BM3D::context.nbSimilarBlocks); 
        hipDeviceSynchronize();
    }
    {
        dim3 numBlocks(BM3D::context.widthBlocksIntern, BM3D::context.widthBlocksIntern);
        dim3 numThreads(1);
        CalculateCoefSimilarBlocks<<<numBlocks,numThreads>>>(BM3D::context.nbSimilarBlocks, BM3D::context.widthBlocksIntern); 
        hipDeviceSynchronize();
    }
    {
        dim3 numBlocks(BM3D::context.widthBlocksIntern, BM3D::context.widthBlocksIntern);
        dim3 numThreads(8, 8);
        Create3DBlocks<<<numBlocks,numThreads>>>(BM3D::context.blocks, BM3D::context.blocks3D, BM3D::context.bmVectors, BM3D::context.widthBlocksIntern); 
        hipDeviceSynchronize(); 
        if(final)
        {
            Create3DBlocks<<<numBlocks,numThreads>>>(BM3D::context.blocksOrig, BM3D::context.blocks3DOrig, BM3D::context.bmVectors, BM3D::context.widthBlocksIntern); 
            hipDeviceSynchronize();
        }
    }
}


__global__
void ShowBlock(int block, int size, double* blocks)
{
    int index = block * 66;
    printf("\n\n");
    for(int i = 0; i < 64; i++) printf("%f, ", blocks[index+i]);
    printf("\nx = %f, y = %f", blocks[index+64], blocks[index+65]);
}

void BM3D::BM3D_ShowBlock(int block)
{
   {
        dim3 numBlocks(1);
        dim3 numThreads(1);
        ShowBlock<<<numBlocks,numThreads>>>(block, BM3D::context.widthBlocks, BM3D::context.blocks); 
        hipDeviceSynchronize();   
   }
}

__global__
void Transform2D_row(double* blocks, int size, double DIVISOR)
{
    int blockIndex = (((blockIdx.y * size) + blockIdx.x) * 66) + (threadIdx.x << 3);
    double inputs[8];
    inputs[0] = blocks[blockIndex];
    inputs[1] = blocks[blockIndex+1];
    inputs[2] = blocks[blockIndex+2];
    inputs[3] = blocks[blockIndex+3];
    inputs[4] = blocks[blockIndex+4];
    inputs[5] = blocks[blockIndex+5];
    inputs[6] = blocks[blockIndex+6];
    inputs[7] = blocks[blockIndex+7];
    Hadamar8(inputs, DIVISOR);
    blocks[blockIndex] = inputs[0];
    blocks[blockIndex+1] = inputs[1];
    blocks[blockIndex+2] = inputs[2];
    blocks[blockIndex+3] = inputs[3];
    blocks[blockIndex+4] = inputs[4];
    blocks[blockIndex+5] = inputs[5];
    blocks[blockIndex+6] = inputs[6];
    blocks[blockIndex+7] = inputs[7];
}

__global__
void Transform2D_col(double* blocks, int size, double DIVISOR)
{
    int blockIndex = (((blockIdx.y * size) + blockIdx.x) * 66) + threadIdx.x;
    double inputs[8];
    int index = blockIndex;
    inputs[0] = blocks[index];
    index += 8;
    inputs[1] = blocks[index];
    index += 8;
    inputs[2] = blocks[index];
    index += 8;
    inputs[3] = blocks[index];
    index += 8;
    inputs[4] = blocks[index];
    index += 8;
    inputs[5] = blocks[index];
    index += 8;
    inputs[6] = blocks[index];
    index += 8;
    inputs[7] = blocks[index];
    Hadamar8(inputs, DIVISOR);
    index = blockIndex;
    blocks[index] = inputs[0];
    index += 8;
    blocks[index] = inputs[1];
    index += 8;
    blocks[index] = inputs[2];
    index += 8;
    blocks[index] = inputs[3];
    index += 8;
    blocks[index] = inputs[4];
    index += 8;
    blocks[index] = inputs[5];
    index += 8;
    blocks[index] = inputs[6];
    index += 8;
    blocks[index] = inputs[7];
}

__global__
void CopyBlocks(double* blocks, double* blocksOrig, int size)
{
    int blockIndex = ((blockIdx.y * size) + blockIdx.x) * 66 + (threadIdx.y << 3) + threadIdx.x;
    blocksOrig[blockIndex] = blocks[blockIndex];
}

void BM3D::BM3D_2DTransform(bool final)
{
   double DIVISOR = sqrt(8);
   {
        dim3 numBlocks(BM3D::context.widthBlocks, BM3D::context.widthBlocks);
        dim3 numThreads(8);
        Transform2D_row<<<numBlocks,numThreads>>>(BM3D::context.blocks, BM3D::context.widthBlocks, DIVISOR); 
        hipDeviceSynchronize();   
   }
   {
        dim3 numBlocks(BM3D::context.widthBlocks, BM3D::context.widthBlocks);
        dim3 numThreads(8);
        Transform2D_col<<<numBlocks,numThreads>>>(BM3D::context.blocks, BM3D::context.widthBlocks, DIVISOR); 
        hipDeviceSynchronize();   
   }
   if(!final)
   {
        dim3 numBlocks(BM3D::context.widthBlocks, BM3D::context.widthBlocks);
        dim3 numThreads(8,8);
        CopyBlocks<<<numBlocks,numThreads>>>(BM3D::context.blocks, BM3D::context.blocksOrig, BM3D::context.widthBlocks); 
        hipDeviceSynchronize();
   }
}

__global__
void ShiftImage(float* originalImage, float* basicImage, int widthOrig, int width, int offset)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int originalImageIndex = (y * widthOrig) + x;
    int basicImageIndex = ((y+offset) * width) + (x+offset);
    basicImage[basicImageIndex] = originalImage[originalImageIndex];
}

__global__
void CreateBlocks(float* basicImage, double* blocks, int size, int width, int pHard)
{
    int blockPixelIndex = (((blockIdx.y * size) + blockIdx.x) * 66) + (threadIdx.y << 3) + threadIdx.x;
    int imgIndex = ((blockIdx.y * pHard + threadIdx.y) * width) + (blockIdx.x * pHard + threadIdx.x);
    blocks[blockPixelIndex] = basicImage[imgIndex];  
}

__global__
void CreateBlocksMap(int* blockMap, int sizeIntern, int size, int pHard)
{
    int blockMapIndex = (((blockIdx.y * sizeIntern) + blockIdx.x) * 1000) + ((threadIdx.y * 10) + threadIdx.x) * 10;
    blockMap[blockMapIndex] = (((blockIdx.y + threadIdx.y) * size) + (blockIdx.x + threadIdx.x)) * 66; //block index
}

__global__
void SetBlockPosition(double* blocks, int size, int pHard)
{
    int blockIndex = ((blockIdx.y * size) + blockIdx.x) * 66;
    blocks[blockIndex+64] = blockIdx.x * pHard;
    blocks[blockIndex+65] = blockIdx.y * pHard;
}

__global__
void InverseShiftImage(float* originalImage, float* basicImage, int widthOrig, int width, int offset)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int originalImageIndex = (y * widthOrig) + x;
    int basicImageIndex = ((y+offset) * width) + (x+offset);
    originalImage[originalImageIndex] = basicImage[basicImageIndex];
}

void BM3D::BM3D_InverseShift()
{
   int offset = 5 * BM3D::context.pHard;
   {
        dim3 numBlocks(BM3D::context.img_widthOrig/8, BM3D::context.img_heightOrig/8);
        dim3 numThreads(8,8);
        InverseShiftImage<<<numBlocks,numThreads>>>(BM3D::context.deviceImage, BM3D::context.basicImage, BM3D::context.img_widthOrig, BM3D::context.img_width, offset); 
        hipDeviceSynchronize();   
   } 
}

void BM3D::BM3D_CreateBlock()
{
    int offset = 5 * BM3D::context.pHard;
    {
        dim3 numBlocks(BM3D::context.img_widthOrig/8, BM3D::context.img_heightOrig/8);
        dim3 numThreads(8,8);
        ShiftImage<<<numBlocks,numThreads>>>(BM3D::context.deviceImage, BM3D::context.basicImage, BM3D::context.img_widthOrig, BM3D::context.img_width, offset); 
        hipDeviceSynchronize();   
    }
    {
        dim3 numBlocks(BM3D::context.widthBlocks, BM3D::context.widthBlocks);
        dim3 numThreads(8,8);
        CreateBlocks<<<numBlocks,numThreads>>>(BM3D::context.basicImage, BM3D::context.blocks, BM3D::context.widthBlocks, BM3D::context.img_width, BM3D::context.pHard); 
        hipDeviceSynchronize();   
    }
    {
        dim3 numBlocks(BM3D::context.widthBlocks, BM3D::context.widthBlocks);
        dim3 numThreads(1);
        SetBlockPosition<<<numBlocks,numThreads>>>(BM3D::context.blocks, BM3D::context.widthBlocks, BM3D::context.pHard); 
        hipDeviceSynchronize();
    }
    {
        dim3 numBlocks(BM3D::context.widthBlocksIntern, BM3D::context.widthBlocksIntern);
        dim3 numThreads(10,10);
        CreateBlocksMap<<<numBlocks,numThreads>>>(BM3D::context.blockMap, BM3D::context.widthBlocksIntern, BM3D::context.widthBlocks, BM3D::context.pHard); 
        hipDeviceSynchronize();   
    }  
}








	
