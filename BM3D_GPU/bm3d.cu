#include "hip/hip_runtime.h"
#include <stdio.h>

#include "bm3d.h"
#include "utilities.h"
#include "timeutil.h"

BM3D::BM3D_Context BM3D::context;

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void BM3D::gpuAssert(hipError_t code, const char *file, int line, bool abort)
{
   if (code != hipSuccess)
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

__global__
void BlocksInitialization(float** blocks, hipfftComplex** dctBlocks, int nHard)
{
    int block = blockIdx.x;
    blocks[block] = (float*)malloc(nHard * nHard * sizeof(float));
    memset(blocks[block], 0, nHard * nHard * sizeof(float));
    dctBlocks[block] = (hipfftComplex*)malloc(nHard * (nHard/2.0) * sizeof(hipfftComplex));
    memset(dctBlocks[block], 0, nHard * (nHard/2.0) * sizeof(hipfftComplex));
}

void BM3D::BM3D_Initialize(BM3D::SourceImage img, int width, int height, int pHard, int nHard)
{
    Timer::startCuda();
    printf("\n--> Execution on Tesla K40c");
    if(hipSuccess != hipSetDevice(0)) printf("\n\tNo device 0 available");

    int sz = 1048576 * 500;
    hipDeviceSetLimit(hipLimitPrintfFifoSize, sz);

    printf("\nBM3D context initialization");
    int widthOffset = width % pHard;
    int heightOffset = height % pHard;
    BM3D::context.img_widthWithBorder = width - widthOffset + nHard;
    BM3D::context.img_heightWithBorder = height - heightOffset + nHard;
    BM3D::context.nbBlocks = ((width - widthOffset) / pHard) * ((height - heightOffset) / pHard);
    BM3D::context.nbBlocksPerLine = ((width - widthOffset) / pHard);

    BM3D::context.img_width = width; 
    BM3D::context.img_height= height;
    BM3D::context.pHard = pHard;
    BM3D::context.nHard = nHard;
    BM3D::context.sourceImage = img;

    gpuErrchk(hipMalloc(&BM3D::context.deviceImage, BM3D::context.img_widthWithBorder * BM3D::context.img_heightWithBorder * sizeof(float)));
    gpuErrchk(hipMemset(BM3D::context.deviceImage, 0, BM3D::context.img_widthWithBorder * BM3D::context.img_heightWithBorder * sizeof(float)));
    gpuErrchk(hipMemcpy(BM3D::context.deviceImage, &img[0], width * height * sizeof(float), hipMemcpyHostToDevice));
   
    //gpuErrchk(hipMalloc(&BM3D::context.deviceBlocks, BM3D::context.nbBlocks * nHard * nHard * sizeof(float)));
    //gpuErrchk(hipMemset(BM3D::context.deviceBlocks, 0, BM3D::context.nbBlocks * nHard * nHard * sizeof(float)));

    hipMalloc((float**)&BM3D::context.deviceBlocks2, BM3D::context.nbBlocks * sizeof(float*));
    hipMalloc((hipfftComplex**)&BM3D::context.deviceBlocksDCT, BM3D::context.nbBlocks * sizeof(hipfftComplex*));
    dim3 threadsPerBlock(1);
    dim3 numBlocks(BM3D::context.nbBlocks);
    BlocksInitialization<<<numBlocks,threadsPerBlock>>>(BM3D::context.deviceBlocks2, BM3D::context.deviceBlocksDCT, nHard);
    hipDeviceSynchronize ();

    Timer::addCuda("Cuda initialization");

    printf("\n\tBorder width (pixel) = %d", (-1 * widthOffset + nHard));
    printf("\n\tBorder height (pixel) = %d", (-1 * heightOffset + nHard));
    printf("\n\tImg width (border) = %d", BM3D::context.img_widthWithBorder);
    printf("\n\tImg height (border) = %d", BM3D::context.img_heightWithBorder);
    printf("\n\tNumber of blocks = %d", BM3D::context.nbBlocks);
    printf("\n\tSize blocks array = %u bytes", (BM3D::context.nbBlocks * nHard * nHard * sizeof(float)));
    printf("\n\tBlock per line= %d", BM3D::context.nbBlocksPerLine);
    printf("\n\tSize block array= %d", BM3D::context.nbBlocks * nHard * nHard);
    printf("\n\tSize Image array= %d", BM3D::context.img_widthWithBorder * BM3D::context.img_heightWithBorder);    
}

void BM3D::BM3D_Run()
{
    printf("\n\nRun BM3D");    
    BM3D_BasicEstimate();

    Timer::showResults();
}

void BM3D::BM3D_BasicEstimate()
{
    printf("\n\tBasic estimates (1 step)");
    BM3D_CreateBlocks();
    //BM3D_2DDCT();
}

void BM3D::BM3D_2DDCT()
{
    Timer::startCuda();
    for(int i=0; i< BM3D::context.nbBlocks; ++i)
    {
        hipfftHandle handle;
        hipfftResult r = hipfftPlan2d(&handle,BM3D::context.nHard,BM3D::context.nHard,HIPFFT_R2C);
        CheckCufftError(r, "hipfftPlan2d");

        r = hipfftExecR2C(handle,BM3D::context.deviceBlocks2[i], BM3D::context.deviceBlocksDCT[i]);
        CheckCufftError(r, "hipfftExecR2C");
        hipDeviceSynchronize ();

        r = hipfftDestroy(handle);
        CheckCufftError(r, "hipfftDestroy");  
    }
    Timer::addCuda("Basic estimate - 2D DCT (Blocks)");
}

__global__
void CreateBlocks_Intern(float* img, float** blocks, const int blocksPerLine, const int pHard, const int nHard, const int width, int sizeBlockArray, int sizeImgArray)
{
    int block = blockIdx.x;
    int x = threadIdx.x;
    int y = threadIdx.y;
    
    int img_y = int(block/(blocksPerLine+1)) * pHard;
    int img_x = (block % blocksPerLine) * pHard;

    //int offsetBlock = (block * nHard * nHard) + (y * nHard) + x;
    int offsetBlock = (y * nHard) + x;
    int offsetImg = ((img_y + y) * width) + img_x + x;

    

    //blocks[offsetBlock] = img[offsetImg];
    blocks[block][offsetBlock] = img[offsetImg];

    bool checkBlockOffset = (offsetBlock >= sizeBlockArray);
    bool checkImgOffset = (offsetImg >= sizeImgArray);
    printf("\nblock %d, line = %d, phard = %d, nhard = %d, width = %d, offsetBlock = %d, offsetImg = %d, img_x = %d, img_y = %d, x = %d, y = %d, b= %s, i=%s, blocks[block][offsetBlock] = %f ", block, blocksPerLine, pHard, nHard, width, offsetBlock, offsetImg, img_x, img_y, x, y, (checkBlockOffset) ? "OUT" : "IN", (checkImgOffset) ? "OUT" : "IN", blocks[block][offsetBlock] );
}

void BM3D::BM3D_CreateBlocks()
{
    Timer::startCuda();
    dim3 threadsPerBlock(BM3D::context.nHard, BM3D::context.nHard);
    dim3 numBlocks(BM3D::context.nbBlocks);
    CreateBlocks_Intern<<<numBlocks,threadsPerBlock>>>( BM3D::context.deviceImage, 
                                                        BM3D::context.deviceBlocks2, 
                                                        BM3D::context.nbBlocksPerLine, 
                                                        BM3D::context.pHard, 
                                                        BM3D::context.nHard, 
                                                        BM3D::context.img_widthWithBorder,
                                                        BM3D::context.nbBlocks * BM3D::context.nHard * BM3D::context.nHard,
                                                        BM3D::context.img_widthWithBorder * BM3D::context.img_heightWithBorder);
    hipDeviceSynchronize ();

    Timer::addCuda("Basic estimate - create blocks");
    //int size = BM3D::context.img_widthWithBorder * BM3D::context.img_heightWithBorder;
    //saveDeviceArray(BM3D::context.deviceImage, size, BM3D::context.img_widthWithBorder, "outputs/img.txt");
    //saveDeviceArray(BM3D::context.deviceBlocks, (BM3D::context.nbBlocks * BM3D::context.nHard * BM3D::context.nHard), (BM3D::context.nHard * BM3D::context.nHard), "outputs/blocks.txt");
}




	
